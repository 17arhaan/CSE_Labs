#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
__global__ void matAddRow(int *a, int *b, int *c, int m, int n) {
    int row = threadIdx.x;
    if (row < m) {
        for (int j = 0; j < n; j++) {
            c[row * n + j] = a[row * n + j] + b[row * n + j];
        }
    }
}
int main() {
    int m, n;
    printf("Enter m and n: ");
    scanf("%d %d", &m, &n);
    int *h_a = (int *)malloc(m * n * sizeof(int));
    int *h_b = (int *)malloc(m * n * sizeof(int));
    int *h_c = (int *)malloc(m * n * sizeof(int));
    printf("Enter mat a: ");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            scanf("%d", &h_a[i * n + j]);
        }
    }
    printf("Enter mat b: ");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            scanf("%d", &h_b[i * n + j]);
        }
    }
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, m * n * sizeof(int));
    hipMalloc(&d_b, m * n * sizeof(int));
    hipMalloc(&d_c, m * n * sizeof(int));
    hipMemcpy(d_a, h_a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, m * n * sizeof(int), hipMemcpyHostToDevice);
    matAddRow<<<1, m>>>(d_a, d_b, d_c, m, n);
    hipMemcpy(h_c, d_c, m * n * sizeof(int), hipMemcpyDeviceToHost);
    printf("Resultant Matrix:\n");
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            printf("%d ", h_c[i * n + j]);
        }
        printf("\n");
    }
    return 0;
}