#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void matMulRow(int *a, int *b, int *c, int wa, int wb) {
    int ridA = threadIdx.x;
    int sum;
    for(int cidB = 0; cidB < wb; cidB++) {
        sum = 0;
        for(int k = 0; k < wa; k++) {
            sum += (a[ridA * wa + k] * b[k * wb + cidB]);
        }
        c[ridA * wb + cidB] = sum;
    }
}
void initializeMatrix(int *matrix, int rows, int cols) {
    printf("Enter the elements of Matrix:\n");
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            scanf("%d",&matrix[i * cols + j]);
        }
    }
}
void printMatrix(int *matrix, int rows, int cols) {
    for(int i = 0; i < rows; i++) {
        for(int j = 0; j < cols; j++) {
            printf("%d ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}
int main() {
    int m, n, p;
    printf("Enter dimensions for matrix multiplication:\n");
    printf("Matrix A rows (m): ");
    scanf("%d", &m);
    printf("Matrix A columns / Matrix B rows (n): ");
    scanf("%d", &n);
    printf("Matrix B columns (p): ");
    scanf("%d", &p);
    int size_a = m * n * sizeof(int);
    int size_b = n * p * sizeof(int);
    int size_c = m * p * sizeof(int);
    int *h_a = (int *)malloc(size_a);
    int *h_b = (int *)malloc(size_b);
    int *h_c = (int *)malloc(size_c);
    printf("Matrix A : \n");
    initializeMatrix(h_a, m, n);
    printf("Matrix B : \n");
    initializeMatrix(h_b, n, p);
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, size_a);
    hipMalloc((void **)&d_b, size_b);
    hipMalloc((void **)&d_c, size_c);
    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    printf("\nUsing one thread per row kernel\n");
    matMulRow<<<1, m>>>(d_a, d_b, d_c, n, p);
    hipMemcpy(h_c, d_c, size_c, hipMemcpyDeviceToHost);
    printf("\nMatrix A (%d x %d):\n", m, n);
    printMatrix(h_a, m, n);
    printf("\nMatrix B (%d x %d):\n", n, p);
    printMatrix(h_b, n, p);
    printf("\nResult Matrix C (%d x %d):\n", m, p);
    printMatrix(h_c, m, p);
    return 0;
}