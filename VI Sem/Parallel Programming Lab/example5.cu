#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <conio.h>

#define N 1024

__global__ void CUDACount(char* A, unsigned int *d_count){
    int i = threadIdx.x;
    if (A[i] == 'a')
        atomicAdd(d_count, 1);
}

int main() {
    char A[N];
    char *d_A;
    unsigned int d_count = 0, *d_result;
    
    printf("Enter a string: ");
    gets(A);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipMalloc((void**)&d_A, strlen(A) * sizeof(char));
    hipMalloc((void**)&d_result, sizeof(unsigned int));
    
    hipMemcpy(d_A, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_result, &d_count, sizeof(unsigned int), hipMemcpyHostToDevice);
    
    hipEventRecord(start, 0);
    CUDACount<<<1, strlen(A)>>>(d_A, d_result);
    hipEventRecord(stop, 0);
    
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    hipMemcpy(&d_count, d_result, sizeof(unsigned int), hipMemcpyDeviceToHost);
    
    printf("Total occurrences of 'a': %d\n", d_count);
    printf("Time Taken: %f ms\n", elapsedTime);
    
    hipFree(d_A);
    hipFree(d_result);
    
    return 0;
}
